#include "hip/hip_runtime.h"
#include "teca_cuda_util.h"

namespace teca_cuda_util
{
// **************************************************************************
int synchronize()
{
    hipError_t ierr = hipSuccess;
    if ((ierr = hipDeviceSynchronize()) != hipSuccess)
    {
        TECA_ERROR("Failed to synchronize CUDA execution. "
            << hipGetErrorString(ierr))
        return -1;
    }
    return 0;
}

// **************************************************************************
int get_local_cuda_devices(MPI_Comm comm, std::deque<int> &local_dev)
{
    hipError_t ierr = hipSuccess;

    // get the number of CUDA GPU's available on this node
    int n_node_dev = 0;
    if ((ierr = hipGetDeviceCount(&n_node_dev)) != hipSuccess)
    {
        TECA_ERROR("Failed to get the number of CUDA devices. "
            << hipGetErrorString(ierr))
        return -1;
    }

    // if there are no GPU's error out
    if (n_node_dev < 1)
    {
        TECA_ERROR("No CUDA devices found")
        return -1;
    }

    // get the number of MPI ranks on this node, and their core id's
#if defined(TECA_HAS_MPI)
    int n_node_ranks = 1;
    int node_rank = 0;

    int is_init = 0;
    MPI_Initialized(&is_init);
    if (is_init)
    {
        // get node local rank and num ranks
        MPI_Comm node_comm;
        MPI_Comm_split_type(comm, MPI_COMM_TYPE_SHARED,
            0, MPI_INFO_NULL, &node_comm);

        MPI_Comm_size(node_comm, &n_node_ranks);
        MPI_Comm_rank(node_comm, &node_rank);

        if (n_node_dev >= n_node_ranks)
        {
            // assign devices evenly between ranks
            int max_dev = n_node_dev - 1;
            int n_per_rank = std::max(n_node_dev / n_node_ranks, 1);
            int n_larger = n_node_dev % n_node_ranks;

            int first_dev = n_per_rank * node_rank + (node_rank < n_larger ? node_rank : n_larger);
            first_dev = std::min(max_dev, first_dev);

            int last_dev = first_dev + n_per_rank - 1 + (node_rank < n_larger ? 1 : 0);
            last_dev = std::min(max_dev, last_dev);

            for (int i = first_dev; i <= last_dev; ++i)
                local_dev.push_back(i);
        }
        else
        {
            // round robbin assignment
            local_dev.push_back( node_rank % n_node_dev );
        }

        MPI_Comm_free(&node_comm);

        return 0;
    }
#endif
    // without MPI this process can use all CUDA devices
    for (int i = 0; i < n_node_dev; ++i)
        local_dev.push_back(i);
    return 0;
}


//-----------------------------------------------------------------------------
int set_device(int device_id)
{
    int n_devices = 0;

    hipError_t ierr = hipGetDeviceCount(&n_devices);
    if (ierr != hipSuccess)
    {
        TECA_ERROR("Failed to get CUDA device count. "
            << hipGetErrorString(ierr))
        return -1;
    }


    if (device_id >= n_devices)
    {
        TECA_ERROR("Attempt to select invalid device "
            << device_id << " of " << n_devices)
        return -1;
    }

    ierr = hipSetDevice(device_id);
    if (ierr)
    {
        TECA_ERROR("Failed to select device " << device_id << ". "
            <<  hipGetErrorString(ierr))
        return -1;
    }

    return 0;
}

// --------------------------------------------------------------------------
int get_launch_props(int device_id,
    int *block_grid_max, int &warp_size,
    int &warps_per_block_max)
{
    hipError_t ierr = hipSuccess;

    if (((ierr = hipDeviceGetAttribute(&block_grid_max[0], hipDeviceAttributeMaxGridDimX, device_id)) != hipSuccess)
        || ((ierr = hipDeviceGetAttribute(&block_grid_max[1], hipDeviceAttributeMaxGridDimY, device_id)) != hipSuccess)
        || ((ierr = hipDeviceGetAttribute(&block_grid_max[2], hipDeviceAttributeMaxGridDimZ, device_id)) != hipSuccess))
    {
        TECA_ERROR("Failed to get CUDA max grid dim. " << hipGetErrorString(ierr))
        return -1;
    }

    if ((ierr = hipDeviceGetAttribute(&warp_size, hipDeviceAttributeWarpSize, device_id)) != hipSuccess)
    {
        TECA_ERROR("Failed to get CUDA warp size. " << hipGetErrorString(ierr))
        return -1;
    }

    int threads_per_block_max = 0;

    if ((ierr = hipDeviceGetAttribute(&threads_per_block_max,
        hipDeviceAttributeMaxThreadsPerBlock, device_id)) != hipSuccess)
    {
        TECA_ERROR("Failed to get CUDA max threads per block. " << hipGetErrorString(ierr))
        return -1;
    }

    warps_per_block_max = threads_per_block_max / warp_size;

    return 0;
}

// --------------------------------------------------------------------------
int partition_thread_blocks(size_t array_size,
    int warps_per_block, int warp_size, int *block_grid_max,
    dim3 &block_grid, int &n_blocks, dim3 &thread_grid)
{
    unsigned long threads_per_block = warps_per_block * warp_size;

    thread_grid.x = threads_per_block;
    thread_grid.y = 1;
    thread_grid.z = 1;

    unsigned long block_size = threads_per_block;
    n_blocks = array_size / block_size;

    if (array_size % block_size)
        ++n_blocks;

    if (n_blocks > block_grid_max[0])
    {
        // multi-d decomp required
        block_grid.x = block_grid_max[0];
        block_grid.y = n_blocks / block_grid_max[0];
        if (n_blocks % block_grid_max[0])
        {
            ++block_grid.y;
        }

        if (block_grid.y > ((unsigned int)block_grid_max[1]))
        {
            // 3d decomp
            unsigned long block_grid_max01 = block_grid_max[0] * block_grid_max[1];
            block_grid.y = block_grid_max[1];
            block_grid.z = n_blocks / block_grid_max01;

            if (n_blocks % block_grid_max01)
                ++block_grid.z;

            if (block_grid.z > ((unsigned int)block_grid_max[2]))
            {
                TECA_ERROR("Too many blocks " << n_blocks << " of size " << block_size
                    << " are required for a grid of (" << block_grid_max[0] << ", "
                    << block_grid_max[1] << ", " << block_grid_max[2]
                    << ") blocks. Hint: increase the number of warps per block.");
                return -1;
            }
        }
        else
        {
            // 2d decomp
            block_grid.z = 1;
        }
    }
    else
    {
        // 1d decomp
        block_grid.x = n_blocks;
        block_grid.y = 1;
        block_grid.z = 1;
    }

    return 0;
}

// --------------------------------------------------------------------------
int partition_thread_blocks(int device_id, size_t array_size,
    int warps_per_block, dim3 &block_grid, int &n_blocks,
    dim3 &thread_grid)
{
    int block_grid_max[3] = {0};
    int warp_size = 0;
    int warps_per_block_max = 0;
    if (get_launch_props(device_id, block_grid_max,
        warp_size, warps_per_block_max))
    {
        TECA_ERROR("Failed to get launch properties")
        return -1;
    }

    return partition_thread_blocks(array_size, warps_per_block,
        warp_size, block_grid_max, block_grid, n_blocks,
        thread_grid);
}

}
